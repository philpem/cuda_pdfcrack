
#include <hip/hip_runtime.h>
#include  <stdio.h>

//  Kernel definition, see also section 2.3 of Nvidia Cuda Programming Guide
__global__  void vecAdd(float* A, float* B, float* C)
{
	// threadIdx.x is a built-in variable  provided by CUDA at runtime
	int i = threadIdx.x;

	A[i] = B[i] = i;
	C[i] = A[i] + B[i];
}

#define  SIZE 10

int main()
{
	int N=SIZE;
	float A[SIZE], B[SIZE], C[SIZE];
	float *devPtrA;
	float *devPtrB;
	float *devPtrC;
	int memsize= SIZE * sizeof(float);

	hipMalloc((void**)&devPtrA, memsize);
	hipMalloc((void**)&devPtrB, memsize);
	hipMalloc((void**)&devPtrC, memsize);
	hipMemcpy(devPtrA, A, memsize,  hipMemcpyHostToDevice);
	hipMemcpy(devPtrB, B, memsize,  hipMemcpyHostToDevice);

	// __global__ functions are called:  Func<<< Dg, Db, Ns  >>>(parameter);
	vecAdd<<<1, N>>>(devPtrA,  devPtrB, devPtrC);
	hipMemcpy(C, devPtrC, memsize,  hipMemcpyDeviceToHost);

	for (int i=0; i<SIZE; i++)
		printf("C[%d]=%f\n",i,C[i]);

	hipFree(devPtrA);
	hipFree(devPtrA);
	hipFree(devPtrA);
}

