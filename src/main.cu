
#include <hip/hip_runtime.h>
#include  <stdio.h>

//  Kernel definition, see also section 2.3 of Nvidia Cuda Programming Guide
__global__  void vecAdd(float* A, float* B, float* C)
{
	// calculate array offset for this thread's global data
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;

	// init a, b
	A[i] = B[i] = i;

	// calculate c
	C[i] = A[i] + B[i];
}

#define  SIZE 10

int main()
{
	int N=SIZE;
	float A[SIZE], B[SIZE], C[SIZE];
	float *devPtrA;
	float *devPtrB;
	float *devPtrC;
	int memsize= SIZE * sizeof(float);

	hipMalloc((void**)&devPtrA, memsize);
	hipMalloc((void**)&devPtrB, memsize);
	hipMalloc((void**)&devPtrC, memsize);
	hipMemcpy(devPtrA, A, memsize,  hipMemcpyHostToDevice);
	hipMemcpy(devPtrB, B, memsize,  hipMemcpyHostToDevice);

	// __global__ functions are called:  Func<<< Dg, Db, Ns  >>>(parameter);
	vecAdd<<<1, N>>>(devPtrA,  devPtrB, devPtrC);
	hipMemcpy(C, devPtrC, memsize,  hipMemcpyDeviceToHost);

	for (int i=0; i<SIZE; i++)
		printf("C[%d]=%f\n",i,C[i]);

	hipFree(devPtrA);
	hipFree(devPtrA);
	hipFree(devPtrA);
}

