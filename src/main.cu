#include "hip/hip_runtime.h"
#include <stdio.h>
#include <ctype.h>
#include "cuda_kernel.h"
#include "password_gen.h"

// number of threads per block
#define THREADSPERBLOCK	10
// number of blocks per grid
#define BLOCKSPERGRID	1

// total number of threads
#define SIZE (THREADSPERBLOCK * BLOCKSPERGRID)

// check parameters
#if (THREADSPERBLOCK > 512)
# error Number of threads per block exceeds maximum permitted by CUDA
#endif

///////////////////////////////////////////////////////////////////

/**
 * @brief Dump the contents of a PDFINFO struct in a human-readable manner
 *
 * @param info The PDFINFO block to be dumped to stdout.
 */
void DumpPDFINFO(PDFINFO_s *p)
{
	// permission flags
	printf("P: %d\n", (int)p->P);

	// crypto key length in bits
	printf("Length: %d\n", p->Length);

	// File ID
	printf("FileID: ");
	for (int i=0; i<16; i++) {
		printf("%02x", p->FileID[i]);
	}
	printf("\n");

	// U
	printf("U: ");
	for (int i=0; i<32; i++) {
		printf("%02x", p->U[i]);
	}
	printf("\n");

	// O
	printf("O: ");
	for (int i=0; i<32; i++) {
		printf("%02x", p->O[i]);
	}
	printf("\n");
}

int ParseCmdline(int argc, char **argv, PDFINFO_s *info)
{
	if (argc < 6) {
		fprintf(stderr, "Syntax: %s P Length FileID U O [Password]\n", argv[0]);
		exit(-1);
	}

	memset(info, 0, sizeof(info));

	// parse command line
	sscanf(argv[1], "%u", &info->P);
	sscanf(argv[2], "%u", &info->Length);

	// parse hexstring argument #1: FileID
	if (strlen(argv[3]) < 32) {
		fprintf(stderr, "ERROR: FileID must be 16 hexpairs (32 characters) in length.\n");
		exit(-1);
	}
	for (int i=0; i<16; i++) {
		char x[3];
		unsigned int y;
		x[0] = tolower(argv[3][(i*2)+0]);
		x[1] = tolower(argv[3][(i*2)+1]);
		x[2] = 0;
		sscanf(x, "%02x", &y);
		info->FileID[i] = y;
	}

	// parse hexstring argument #2: U
	if (strlen(argv[4]) < 64) {
		fprintf(stderr, "ERROR: U must be 32 hexpairs (64 characters) in length.\n");
		exit(-1);
	}
	for (int i=0; i<32; i++) {
		char x[3];
		unsigned int y;
		x[0] = tolower(argv[4][(i*2)+0]);
		x[1] = tolower(argv[4][(i*2)+1]);
		x[2] = 0;
		sscanf(x, "%02x", &y);
		info->U[i] = y;
	}

	// parse hexstring argument #3: O
	if (strlen(argv[5]) < 64) {
		fprintf(stderr, "ERROR: O must be 32 hexpairs (64 characters) in length.\n");
		exit(-1);
	}
	for (int i=0; i<32; i++) {
		char x[3];
		unsigned int y;
		x[0] = tolower(argv[5][(i*2)+0]);
		x[1] = tolower(argv[5][(i*2)+1]);
		x[2] = 0;
		sscanf(x, "%02x", &y);
		info->O[i] = y;
	}

	return 0;
}

///////////////////////////////////////////////////////////////////
void hex_dump(void *data, int size)
{
    /* dumps size bytes of *data to stdout. Looks like:
     * [0000] 75 6E 6B 6E 6F 77 6E 20
     *                  30 FF 00 00 00 00 39 00 unknown 0.....9.
     * (in a single line of course)
     */

    unsigned char *p = (unsigned char *)data;
    unsigned long addr = 0;
    unsigned char c;
    int n;
    char bytestr[4] = {0};
    char addrstr[10] = {0};
    char hexstr[ 16*3 + 5] = {0};
    char charstr[16*1 + 5] = {0};
    for(n=1;n<=size;n++) {
        if (n%16 == 1) {
            /* store address for this line */
            snprintf(addrstr, sizeof(addrstr), "%.4lX",
               addr);
        }
            
        c = *p;
        if (isalnum(c) == 0) {
            c = '.';
        }

        /* store hex str (for left side) */
        snprintf(bytestr, sizeof(bytestr), "%02X ", *p);
        strncat(hexstr, bytestr, sizeof(hexstr)-strlen(hexstr)-1);

        /* store char str (for right side) */
        snprintf(bytestr, sizeof(bytestr), "%c", c);
        strncat(charstr, bytestr, sizeof(charstr)-strlen(charstr)-1);

        if(n%16 == 0) { 
            /* line completed */
            printf("[%4.4s]   %-50.50s  %s\n", addrstr, hexstr, charstr);
            hexstr[0] = 0;
            charstr[0] = 0;
        } else if(n%8 == 0) {
            /* half line: add whitespaces */
            strncat(hexstr, "  ", sizeof(hexstr)-strlen(hexstr)-1);
            strncat(charstr, " ", sizeof(charstr)-strlen(charstr)-1);
        }
        p++; /* next byte */
		addr++; /* increment address */
    }

    if (strlen(hexstr) > 0) {
        /* print rest of buffer if not empty */
        printf("[%4.4s]   %-50.50s  %s\n", addrstr, hexstr, charstr);
    }
}

int main(int argc, char **argv)
{
	ComputeBlock cb[SIZE];
	ComputeBlock *devPtrCb;
	int memsize = SIZE * sizeof(ComputeBlock);
	PDFINFO_s pdfinfo_loc;

	// Parse the command line
	ParseCmdline(argc, argv, &pdfinfo_loc);

	DumpPDFINFO(&pdfinfo_loc);

	// allocate GPU memory for the calculations
	hipMalloc((void**)&devPtrCb, memsize);

	// initialise input array
	for (int i=0; i<SIZE; i++) {
		cb[i].pwlen = 0;
		cb[i].match = 99;		// system error
	}

	// generate passwords
	int len=1;
	int counter[32];
	char str[33];
	password_init(32, counter, str);
	do {
		for (int i=0; i<SIZE; i++) {
			str[len]='\0';
			printf("%s\n", str);
			if (password_next(len, counter, str)) {
				password_init(len+1, counter, str);
				len++;
			}
			if (len > 2) break;
		}
	} while (len <= 2);

return;

/*
	// create some input data
	for (int i=0; i<SIZE; i++) {
		const char *PASSWD="usea";
		strcpy(cb[i].password, PASSWD);
		cb[i].password[strlen(PASSWD)-1] += i;
		cb[i].pwlen = strlen(PASSWD);
		cb[i].match = 99;
	}
*/
	// copy input data to the graphics chip
	hipMemcpy(devPtrCb, cb, memsize, hipMemcpyHostToDevice);

	// Copy PDFINFO block from CPU --> GPU "constant RAM" space
	LoadPdfInfo(&pdfinfo_loc);

	// __global__ functions are called:  Func<<< Dg, Db, Ns  >>>(parameter);
	ComputeKernel <<< BLOCKSPERGRID, THREADSPERBLOCK >>> (devPtrCb);

	// copy result from GPU to local CPU RAM
	hipMemcpy(cb, devPtrCb, memsize, hipMemcpyDeviceToHost);

	for (int i=0; i<SIZE; i++) {
		if (cb[i].pwlen == 0) break;
		cb[i].password[cb[i].pwlen] = '\0';
		printf("%3d\t%s\t%s\n", i, cb[i].password, ((cb[i].match == 99) ? "SysError" : (cb[i].match ? "MATCH" : "fail")));
	}

	// free GPU memory
	hipFree(devPtrCb);
}

